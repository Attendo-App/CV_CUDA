#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Sobel_CUDA.h"
#include <iostream>
#include <stdio.h>

__global__ void Gaussian_Blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image);

unsigned char* Gaussian_Blur_CUDA(unsigned char* Input_Image, int Height, int Width) {

    unsigned char* Dev_Input_Image = NULL;
    //allocate the memory in gpu
    hipMalloc((void**)&Dev_Input_Image, Height * Width * sizeof(unsigned char));
    //copy image to gpu
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * sizeof(unsigned char), hipMemcpyHostToDevice);

    //allocating space for output image
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Output_Image, Height * Width * sizeof(unsigned char));

    //specifying grid and block size.
    //since there doesnt need to be any inter-thread communication, we keep block size (1,1)
    dim3 Grid_Image(Height , Width );
    dim3 Block_size(1, 1);
    
    size_t shm_size = 4 * sizeof(unsigned long long);
    Gaussian_Blur_CUDA_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image);

    unsigned char* result = (unsigned char*)malloc(sizeof(unsigned char*) * Height * Width);
    //copy processed data back to cpu from gpu
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));


    //free gpu mempry
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    return result;
}

__global__ void Gaussian_Blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image)
{
    int i = blockIdx.x;
    int j = blockIdx.y;

    int height = gridDim.x;
    int width = gridDim.y;

    float val = 0;
  
    for (int k = -1; k <= 1; k++)
    {
        for (int l = -1; l <= 1; l++)
        {
            int x = i + k;
            int y = j + l;
            int z = abs(l) + abs(k);
            printf(l + "Hello" + k);
            if (x >= 0 && y >= 0 && x < height && y < width)
            {
                if (z == 1)
                {
                    val += 24 * Dev_Input_Image[x * width + y];
                }
                else if (z == 2)
                {
                    if (l == 0 || k ==0 )
                    {
                        val += 6 * Dev_Input_Image[x * width + y];
                    }
                    else
                    {
                        val += 16 * Dev_Input_Image[x * width + y];
                    }
                }
                else if (z == 3)
                {
                    val += 4 *  Dev_Input_Image[x * width + y];
                }
                else if (z == 4)
                {
                    val += Dev_Input_Image[x * width + y];
                }
                else
                {
                    val += 36 * Dev_Input_Image[x * width + y];
                }
            }
        }
    }

    val /= 256;

    Dev_Output_Image[(i * width) + j] = val;
}
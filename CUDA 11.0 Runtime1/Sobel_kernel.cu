#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Sobel_CUDA.h"
#include <iostream>
#include <stdio.h>

__global__ void Sobel_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image);

unsigned char* Sobel_CUDA(unsigned char* Input_Image, int Height, int Width) { 
    
    unsigned char* Dev_Input_Image = NULL;
	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * sizeof(unsigned char));
    //copy image to gpu
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * sizeof(unsigned char), hipMemcpyHostToDevice);

    //allocating space for output image
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Output_Image, Height * Width * sizeof(unsigned char));

    //specifying grid and block size.
    //since there doesnt need to be any inter-thread communication, we keep block size (1,1)
    dim3 Grid_Image(Height - 2, Width - 2);
    dim3 Block_size(1, 1);

    size_t shm_size = 4 * sizeof(unsigned long long);
	Sobel_CUDA_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image);

    unsigned char* result = (unsigned char*)malloc(sizeof(unsigned char*) * Height * Width);
	//copy processed data back to cpu from gpu
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));
	

	//free gpu mempry
	hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    return result;
}

__global__ void Sobel_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image)
{  

    int i = blockIdx.x + 1;
    int j = blockIdx.y + 1;

    int horizontalDiff = 0;
    int verticalDiff = 0;

    int width = gridDim.y + 2;

    horizontalDiff = Dev_Input_Image[(i - 1) * width + (j + 1) ] - Dev_Input_Image[(i - 1) * width + (j - 1)];
    horizontalDiff += 2 * (Dev_Input_Image[(i) *width + (j + 1)] - Dev_Input_Image[(i) *width + (j - 1)]);
    horizontalDiff = Dev_Input_Image[(i + 1) * width + (j + 1)] - Dev_Input_Image[(i+1) * width + (j - 1) ];

    verticalDiff = -Dev_Input_Image[(i - 1) * width + (j - 1)] - Dev_Input_Image[(i - 1) * width + (j + 1)];
    verticalDiff += Dev_Input_Image[(i + 1) * width + (j + 1)] + Dev_Input_Image[(i + 1) * width + (j - 1)];
    verticalDiff += 2 * (Dev_Input_Image[(i+1) * width + (j)] - Dev_Input_Image[(i - 1) * width + (j)]);

    Dev_Output_Image[(i - 1) * width + (j - 1)] = sqrt((float)(horizontalDiff * horizontalDiff + verticalDiff * verticalDiff))/4;
}
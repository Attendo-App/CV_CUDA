#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Sobel_CUDA.h"
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void Salt_Pepper_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image, hiprandState_t* state);
__global__ void init(hiprandState_t* state, int seed);

unsigned char* Salt_Pepper(unsigned char* Input_Image, int Height, int Width) {

    hiprandState_t* Dev_Rand_State;
    hipMalloc((void**)&Dev_Rand_State, sizeof(hiprandState_t));
    init << <1, 1 >> > (Dev_Rand_State, time(0));

    unsigned char* Dev_Input_Image = NULL;
    //allocate the memory in gpu
    hipMalloc((void**)&Dev_Input_Image, Height * Width * sizeof(unsigned char));
    //copy image to gpu
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * sizeof(unsigned char), hipMemcpyHostToDevice);

    //allocating space for output image
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Output_Image, Height * Width * sizeof(unsigned char));

    //specifying grid and block size.
    //since there doesnt need to be any inter-thread communication, we keep block size (1,1)
    dim3 Grid_Image(Height, Width);
    dim3 Block_size(1, 1);

    size_t shm_size = 4 * sizeof(unsigned long long);
    Salt_Pepper_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image, Dev_Rand_State);

    unsigned char* result = (unsigned char*)malloc(sizeof(unsigned char*) * Height * Width);
    //copy processed data back to cpu from gpu
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));


    //free gpu mempry
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    return result;
}

__global__ void Salt_Pepper_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image, hiprandState_t* state)
{

    int i = blockIdx.x;
    int j = blockIdx.y;

    int width = gridDim.y;
    
    Dev_Output_Image[(i)*width + (j)] = Dev_Input_Image[(i)*width + (j)];
    int x, y, z;
    x = hiprand(state) % 2;
    y = hiprand(state) % 2;
    z = hiprand(state) % 2;
    if (x == 1 && y == 1 && z == 1)
    {
        Dev_Output_Image[(i)*width + (j)] = 255;
    }
    else if (x == 0 && y == 0 && z == 1)
    {
        Dev_Output_Image[(i)*width + (j)] = 0;
    }
}

__global__ void init(hiprandState_t* state, int seed)
{
    /* we have to initialize the state */
    hiprand_init(0, 0, 0, state);
}
#include "hip/hip_runtime.h"
/*#include "hip/hip_runtime.h"
#include ""
#include "Sobel_CUDA.h"
#include <iostream>
#include <stdio.h>*/
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "Driver.h"
#include "Sobel_CUDA.h"
#include "Gaussian_Blur.h"
#include "Noise_Addition_CUDA.h"
#include "Gaussian_Blur_Seperated.h"
#include "Mean_Blur_Seperated.h"
#include "Canny_CUDA.h"
#include "sharpen_CUDA.h"
#include "bokeh_blur.h"
using namespace cv;
using namespace std;

__global__ void Bokeh_Blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image , unsigned char* image,  int h,  int w);
unsigned char* Bokeh_Blur_CUDA(unsigned char* Input_Image , int Height, int Width , unsigned char* Image, int h, int w) {

    unsigned char* Dev_Input_Image = NULL;
    unsigned char* image = NULL;
    //allocate the memory in gpu
    hipMalloc((void**)&Dev_Input_Image, Height * Width * sizeof(unsigned char));
    hipMalloc((void**)&image, h * w * sizeof(unsigned char));
    //copy image to gpu
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(image, Image, h * w * sizeof(unsigned char), hipMemcpyHostToDevice);
    //allocating space for output image
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Output_Image, Height * Width * sizeof(unsigned char));

    //specifying grid and block size.
    //since there doesnt need to be any inter-thread communication, we keep block size (1,1)
    dim3 Grid_Image(Height, Width);
    dim3 Block_size(1, 1);

    size_t shm_size = 4 * sizeof(unsigned long long);
    //Bokeh_Blur_CUDA_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image);
    Bokeh_Blur_CUDA_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image , image, h, w);

    unsigned char* result = (unsigned char*)malloc(sizeof(unsigned char*) * Height * Width);
    //copy processed data back to cpu from gpu
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));


    //free gpu mempry
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
    hipFree(image);

    return result;
}

__global__ void Bokeh_Blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image , unsigned char* image,  int h,  int w)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    
    int height = gridDim.x;
    int width = gridDim.y;
    float val = 0;
    float total = 0;

    for (int k = 0; k < h; k++)
    {
        for (int l = 0; l < w; l++)
        {
            int x = i + k;
            int y = j + l;
            if (x >= 0 && y >= 0 && x < height && y < width)
            {
                    val += Dev_Input_Image[x * width + y] * image[k * h + l];
                    total += image[k * h + l];
            }
        }
    }
    val/=total;

    Dev_Output_Image[(i * width) + j] = val;
}


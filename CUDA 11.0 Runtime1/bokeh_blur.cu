#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Sobel_CUDA.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;

__global__ void bokeh_blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image);

unsigned char* bokeh_blur_CUDA(unsigned char* Input_Image, int Height, int Width) {

    unsigned char* Dev_Input_Image = NULL;
    //allocate the memory in gpu
    hipMalloc((void**)&Dev_Input_Image, Height * Width * sizeof(unsigned char));
    //copy image to gpu
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * sizeof(unsigned char), hipMemcpyHostToDevice);

    //allocating space for output image
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Output_Image, Height * Width * sizeof(unsigned char));

    //specifying grid and block size.
    //since there doesnt need to be any inter-thread communication, we keep block size (1,1)
    dim3 Grid_Image(Height, Width);
    dim3 Block_size(1, 1);

    size_t shm_size = 4 * sizeof(unsigned long long);
    bokeh_blur_CUDA_Kernel << <Grid_Image, Block_size, shm_size >> > (Dev_Input_Image, Dev_Output_Image);

    unsigned char* result = (unsigned char*)malloc(sizeof(unsigned char*) * Height * Width);
    //copy processed data back to cpu from gpu
    hipMemcpy(Input_Image, Dev_Output_Image, Height * Width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));


    //free gpu mempry
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    return result;
}

__global__ void bokeh_blur_CUDA_Kernel(unsigned char* Dev_Input_Image, unsigned char* Dev_Output_Image)
{

    int i = blockIdx.x;
    int j = blockIdx.y;

    int horizontalDiff = 0;
    int verticalDiff = 0;

    int width = gridDim.y + 2;

    horizontalDiff = Dev_Input_Image[(i - 1) * width + (j + 1)] - Dev_Input_Image[(i - 1) * width + (j - 1)];
    horizontalDiff += 2 * (Dev_Input_Image[(i)*width + (j + 1)] - Dev_Input_Image[(i)*width + (j - 1)]);
    horizontalDiff = Dev_Input_Image[(i + 1) * width + (j + 1)] - Dev_Input_Image[(i + 1) * width + (j - 1)];

    verticalDiff = -Dev_Input_Image[(i - 1) * width + (j - 1)] - Dev_Input_Image[(i - 1) * width + (j + 1)];
    verticalDiff += Dev_Input_Image[(i + 1) * width + (j + 1)] + Dev_Input_Image[(i + 1) * width + (j - 1)];
    verticalDiff += 2 * (Dev_Input_Image[(i + 1) * width + (j)] - Dev_Input_Image[(i - 1) * width + (j)]);

    Dev_Output_Image[(i - 1) * width + (j - 1)] = sqrt((float)(horizontalDiff * horizontalDiff + verticalDiff * verticalDiff));
}